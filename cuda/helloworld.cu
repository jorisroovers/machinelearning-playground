#include "hip/hip_runtime.h"
// This code is from this tutorial:
// https://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/
// This is a so-called SAXPY example. SAXPY stands for "Single-precision A*X Plus Y", and is a good hello world example
// for parallel computation.

#include <stdio.h>

// Functions starting wit __global__ are "device" functions, or "kernels" that run on the GPU.
// (sometimes they're also referred to as (compute) shaders as a reference to graphics shaders, the name of these
// kernels when used for in a graphics context).
// GPU have a large number of cores (sometimes several thousands) that run these kernels in parallel.
// The cores are are grouped in blocks of dimensions NxN.
// This particular kernel will do a single element multiplication in a vector multiplication. The GPU will invoke
// this kernel in every thread, and will automatically replace the magic variables blockIdx, blockDim and threadIdx
// with the corresponding block number, dimensions and thread ID (within the block). This way, you can figure out
// what part of the calculation to do within a certain kernel instantiation.
__global__ void saxpy(int n, float a, float *x, float *y)
{
  // Determine which calculation to perform in this particular thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

// Normal functions are referred to as "host" functions/code and run on the CPU as usual
int main(void)
{
  // Allocate 2 vectors, x and y with 1 million floats (=single precision) each
  int N = 1 << 20;          // 2^20 ~= 1 million
  float *x, *y, *d_x, *d_y; // d_x -> "device x", d_y -> "device y"
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  // Allocate the memory on the GPUs
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  // set all values in x to 1, all values in y to 2
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // copy the values from the host to the GPU
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // Note that triple angular brackets, that's specific to CUDA
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  // We stored the result in Y on the device, copy it back to the host
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  // clean up :)
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}